#include "hip/hip_runtime.h"
#include "config.h"

__device__ void compute_system(IN double h, IN double x, IN double y[NUM_EQ],
                               IN double g[NUM_PAR],
                               OUT double dy[NUM_DIFF][NUM_EQ]) {
  double ig[NUM_DIFF][NUM_INTPAR];

  ig[0][0] = -y[0]*g[12]+y[3]*g[9];
  ig[0][1] =  y[4]*g[10]-g[13]*y[1];
  ig[0][2] =  g[11]*y[5]-y[2]*g[14];
  ig[0][3] = g[1]/pow( pow(1.0/g[17]*y[0],g[25])+1.0,2.0)*g[25]*pow(1.0/g[17]*y[0], g[25]-1.0)/g[17];
  ig[0][4] = 0.0;
  ig[0][5] = 0.0;
  ig[0][6] = 0.0;
  ig[0][7] = pow(1.0/g[21]*y[1], g[25]-1.0)*g[25]*g[2]/g[21]/pow( pow(1.0/g[21]*y[1],g[25])+1.0,2.0);
  ig[0][8] = 0.0;
  ig[0][9] = 0.0;
  ig[0][10] = 0.0;
  ig[0][11] = g[0]*pow(y[2]/g[22], g[25]-1.0)*g[25]/g[22]/pow( pow(y[2]/g[22],g[25])+1.0,2.0);
  ig[0][12] = -1.0/( ig[0][3]+1.0)*ig[0][3]+1.0;
  ig[0][13] = 0.0;
  ig[0][14] = 0.0;
  ig[0][15] = 0.0;
  ig[0][16] = -ig[0][7]/( ig[0][7]+1.0)+1.0;
  ig[0][17] = 0.0;
  ig[0][18] = 0.0;
  ig[0][19] = 0.0;
  ig[0][20] = -1.0/( ig[0][11]+1.0)*ig[0][11]+1.0;
  dy[0][0] = ig[0][0]*ig[0][12];
  dy[0][1] = ig[0][1]*ig[0][16];
  dy[0][2] = ig[0][2]*ig[0][20];
  dy[0][3] =  y[3]*g[6]-( g[15]*pow(y[2]/g[22],g[25])/( pow(y[2]/g[22],g[25])+1.0)+1.0)*g[0]*g[3];
  dy[0][4] = -g[1]*( g[15]/( pow(1.0/g[17]*y[0],g[25])+1.0)*pow(1.0/g[17]*y[0],g[25])+1.0)*g[4]+g[7]*y[4];
  dy[0][5] = -( g[15]*pow(1.0/g[21]*y[1],g[25])/( pow(1.0/g[21]*y[1],g[25])+1.0)+1.0)*g[2]*g[5]+g[8]*y[5];
  ig[1][0] =  g[9]*dy[0][3]-dy[0][0]*g[12];
  ig[1][1] =  dy[0][4]*g[10]-g[13]*dy[0][1];
  ig[1][2] = -dy[0][2]*g[14]+dy[0][5]*g[11];
  ig[1][3] =  g[1]/pow( pow(1.0/g[17]*y[0],g[25])+1.0,2.0)*dy[0][0]*g[25]*( g[25]-1.0)*pow(1.0/g[17]*y[0], g[25]-1.0)/g[17]/y[0]+-2.0*g[1]/pow( pow(1.0/g[17]*y[0],g[25])+1.0,3.0)*dy[0][0]*pow(g[25],2.0)*pow(1.0/g[17]*y[0], g[25]-1.0)/g[17]*pow(1.0/g[17]*y[0],g[25])/y[0];
  ig[1][4] = 0.0;
  ig[1][5] = 0.0;
  ig[1][6] = 0.0;
  ig[1][7] =  pow(1.0/g[21]*y[1], g[25]-1.0)*g[25]*( g[25]-1.0)*g[2]*dy[0][1]/g[21]/pow( pow(1.0/g[21]*y[1],g[25])+1.0,2.0)/y[1]+-2.0*pow(1.0/g[21]*y[1], g[25]-1.0)*pow(1.0/g[21]*y[1],g[25])*pow(g[25],2.0)*g[2]*dy[0][1]/g[21]/pow( pow(1.0/g[21]*y[1],g[25])+1.0,3.0)/y[1];
  ig[1][8] = 0.0;
  ig[1][9] = 0.0;
  ig[1][10] = 0.0;
  ig[1][11] =  -2.0*1.0/y[2]*g[0]*pow(y[2]/g[22], g[25]-1.0)*pow(y[2]/g[22],g[25])*pow(g[25],2.0)*dy[0][2]/g[22]/pow( pow(y[2]/g[22],g[25])+1.0,3.0)+1.0/y[2]*g[0]*pow(y[2]/g[22], g[25]-1.0)*g[25]*( g[25]-1.0)*dy[0][2]/g[22]/pow( pow(y[2]/g[22],g[25])+1.0,2.0);
  ig[1][12] =  ig[1][3]/pow( ig[0][3]+1.0,2.0)*ig[0][3]-ig[1][3]/( ig[0][3]+1.0);
  ig[1][13] = 0.0;
  ig[1][14] = 0.0;
  ig[1][15] = 0.0;
  ig[1][16] = -ig[1][7]/( ig[0][7]+1.0)+ig[1][7]*ig[0][7]/pow( ig[0][7]+1.0,2.0);
  ig[1][17] = 0.0;
  ig[1][18] = 0.0;
  ig[1][19] = 0.0;
  ig[1][20] = -ig[1][11]/( ig[0][11]+1.0)+ig[1][11]/pow( ig[0][11]+1.0,2.0)*ig[0][11];
  dy[1][0] =  ig[1][12]*ig[0][0]+ig[1][0]*ig[0][12];
  dy[1][1] =  ig[1][1]*ig[0][16]+ig[1][16]*ig[0][1];
  dy[1][2] =  ig[1][2]*ig[0][20]+ig[1][20]*ig[0][2];
  dy[1][3] = -( 1.0/y[2]*g[15]*pow(y[2]/g[22],g[25])*g[25]*dy[0][2]/( pow(y[2]/g[22],g[25])+1.0)-1.0/y[2]*g[15]*pow(pow(y[2]/g[22],g[25]),2.0)*g[25]*dy[0][2]/pow( pow(y[2]/g[22],g[25])+1.0,2.0))*g[0]*g[3]+g[6]*dy[0][3];
  dy[1][4] =  dy[0][4]*g[7]-g[1]*g[4]*( g[15]/( pow(1.0/g[17]*y[0],g[25])+1.0)*dy[0][0]*g[25]*pow(1.0/g[17]*y[0],g[25])/y[0]-g[15]/pow( pow(1.0/g[17]*y[0],g[25])+1.0,2.0)*dy[0][0]*g[25]*pow(pow(1.0/g[17]*y[0],g[25]),2.0)/y[0]);
  dy[1][5] =  g[8]*dy[0][5]+g[2]*( g[15]*pow(pow(1.0/g[21]*y[1],g[25]),2.0)*g[25]*dy[0][1]/pow( pow(1.0/g[21]*y[1],g[25])+1.0,2.0)/y[1]-g[15]*pow(1.0/g[21]*y[1],g[25])*g[25]*dy[0][1]/( pow(1.0/g[21]*y[1],g[25])+1.0)/y[1])*g[5];


  double hx = h;
  for (uint16_t i = 0; i < NUM_DIFF; ++i) {
    for (uint16_t j = 0; j < NUM_EQ; ++j) {
      dy[i][j] *= hx;
    }
    hx *= h;
  }
}

__global__ void solver_main(IN double x, IN_OUT double y_global[Y_GLOBAL_SIZE],
                            IN double g_global[G_GLOBAL_SIZE],
                            IN_OUT uint64_t f_cnts_global[NUM_SYS]) {
  const uint64_t sys_index =
      (uint64_t)blockDim.x * blockIdx.x + (uint64_t)threadIdx.x;

  if (sys_index < NUM_SYS) {
    uint64_t f_cnt = 0;
    double y[NUM_EQ], g[NUM_PAR];

    for (uint16_t i = 0; i < NUM_EQ; ++i) {
      y[i] = y_global[sys_index + NUM_SYS * i];
    }
    for (uint16_t i = 0; i < NUM_PAR; ++i) {
      g[i] = g_global[sys_index + NUM_SYS * i];
    }

    constexpr double a[NUM_STAGE][NUM_STAGE][NUM_DIFF] = {{},{{(3.0/11.0),(9.0/242.0)}},{{(18.0/25.0),-(9.0/15625.0)},{0.0,(4059.0/15625.0)}}};

    constexpr double b[NUM_STAGE][NUM_DIFF] = {{1.0,(53.0/648.0)},{0.0,(1331.0/4428.0)},{0.0,(3125.0/26568.0)}};

    constexpr double eb[NUM_STAGE][NUM_DIFF] = {{(783089.0/1417500.0),(5989.0/157500.0)},{(3115871.0/9686250.0),(28919.0/157500.0)},{(11705.0/92988.0),(1.0/10.0)}};

    constexpr double c[NUM_STAGE] = {0.0,(3.0/11.0),(18.0/25.0)};

    double x_curr = x, x_end = x + STEP_SIZE;
    double h = 0.5 * STEP_SIZE;

    while (x_curr < x_end) {
      f_cnt += NUM_STAGE;
      double y_temp[NUM_EQ];

      double k[NUM_STAGE][NUM_DIFF][NUM_EQ];
      compute_system(h, x_curr + h * c[0], y, g, k[0]);

      for (uint16_t stage = 1; stage < NUM_STAGE; ++stage) {
        for (uint16_t eq = 0; eq < NUM_EQ; ++eq) {
          y_temp[eq] = y[eq];
          for (uint16_t i = 0; i < stage; ++i) {
            for (uint16_t j = 0; j < NUM_DIFF; ++j) {
              y_temp[eq] += a[stage][i][j] * k[i][j][eq];
            }
          }
        }
        compute_system(h, x_curr + h * c[stage], y_temp, g, k[stage]);
      }

      for (uint16_t eq = 0; eq < NUM_EQ; ++eq) {
        y_temp[eq] = y[eq];
        for (uint16_t i = 0; i < NUM_STAGE; ++i) {
          for (uint16_t j = 0; j < NUM_DIFF; ++j) {
            y_temp[eq] += b[i][j] * k[i][j][eq];
          }
        }
      }

      double err = 0.0;
      bool is_nan = false;
      for (uint16_t eq = 0; eq < NUM_EQ; ++eq) {
        double ey = y[eq];
        for (uint16_t i = 0; i < NUM_STAGE; ++i) {
          for (uint16_t j = 0; j < NUM_DIFF; ++j) {
            ey += eb[i][j] * k[i][j][eq];
          }
        }
        double err_curr = y_temp[eq] - ey;

        err = fmax(err,
                   fabs(err_curr / (fabs(y[eq]) + fabs(k[0][0][eq]) + TINY)));
        is_nan |= isnan(err) | isnan(err_curr);
      }
      err /= EPS;

      if (is_nan) {
        h *= P1;
      } else if (err > 1.0) {
        h *= fmax(SAFETY * pow(err, PSHRINK), P1);
      } else {
        x_curr += h;
        for (uint16_t i = 0; i < NUM_EQ; ++i) {
          y[i] = y_temp[i];
        }
        h *= fmin(SAFETY * pow(err, PGROW), P2);
      }

      h = fmin(h, x_end - x_curr);
    }

    f_cnts_global[sys_index] += f_cnt;
    for (uint16_t i = 0; i < NUM_EQ; ++i) {
      y_global[sys_index + NUM_SYS * i] = y[i];
    }
  }
}